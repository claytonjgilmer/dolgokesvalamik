#include "hip/hip_runtime.h"
// incrementArray.cu
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define TYPE float
typedef struct { float x; float y[3];} vec2;
//typedef struct { float x;} vec2;

void incrementArrayOnHost(vec2 *a, int N)
{
	int i;
	for (i=0; i < N; i++)
	{
		a[i].x+=1.f;
//		a[i].y+=1.f;
	}
}
__global__ void incrementArrayOnDevice(vec2 *a, int N, int LoopC)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
//if (idx<N) a[idx] = a[idx]+1.f;
	for (int n=0; n<LoopC; ++n)
		a[idx].x+=1.f;
//	a[idx].y+=1.f;
}
int main(void)
{
vec2 *a_h, *b_h;           // pointers to host memory
vec2 *a_d;                 // pointer to device memory
int i, N = 256*256;
int loopc=100000;
size_t size = N*sizeof(vec2);
// allocate arrays on host
a_h = (vec2*)malloc(size);
b_h = (vec2*)malloc(size);
// allocate array on device 
hipMalloc((void**) &a_d, size);
// initialization of host data

for (i=0; i<N; i++)
{
	a_h[i].x = (float)i;
//	a_h[i].y = -(float)i;
}

// copy data from host to device
hipMemcpy(a_d, a_h, sizeof(vec2)*N, hipMemcpyHostToDevice);
printf("do calculation on host\n");

for (int n=0; n<loopc; ++n)
	incrementArrayOnHost(a_h, N);
printf("do calculation on device\n");
// Part 1 of 2. Compute execution configuration
int blockSize = 128;
int nBlocks = N/blockSize + (N%blockSize == 0?0:1);
// Part 2 of 2. Call incrementArrayOnDevice kernel 

//for (int n=0; n<loopc; ++n)
incrementArrayOnDevice <<< nBlocks, blockSize >>> (a_d, N,loopc);
hipDeviceSynchronize();
	
printf("Retrieve result from device and store in b_h\n");
hipMemcpy(b_h, a_d, sizeof(vec2)*N, hipMemcpyDeviceToHost);
printf("check results\n");

int diffc=0;
for (i=0; i<N; i++)
{
	if (a_h[i].x != b_h[i].x)// || a_h[i].y != b_h[i].y)
		++diffc;
}
	
printf("diffcount:%d\n",diffc);
printf("cleanup\n");
free(a_h); free(b_h); hipFree(a_d); 
}
